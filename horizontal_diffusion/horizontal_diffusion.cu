#include "hip/hip_runtime.h"
#include "horizontal_diffusion.h"
#include "../repository.hpp"
#include "../utils.hpp"
#include "horizontal_diffusion_reference.hpp"
#include "../functions.hpp"


class BlockSyncer {
private:
    bool dir;
    volatile unsigned int* lock_var;

public:
    __device__ BlockSyncer(volatile unsigned int* lock_var)
        : dir(false)
        , lock_var(lock_var){};
    __device__ void sync()
    {
        if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            dir = !dir;
            atomicAdd((unsigned int*)lock_var, dir ? 1 : -1);
            while(gridDim.x * gridDim.y * gridDim.z * dir != *lock_var)
                ;
        }
        __syncthreads();
    }
};

#define BLOCK_X_SIZE 32
#define BLOCK_Y_SIZE 8

#define HALO_BLOCK_X_MINUS 1
#define HALO_BLOCK_X_PLUS 1

#define HALO_BLOCK_Y_MINUS 1
#define HALO_BLOCK_Y_PLUS 1

#define PADDED_BOUNDARY 1

inline __device__ unsigned int cache_index(const int ipos, const int jpos) {
    return (ipos + PADDED_BOUNDARY) +
           (jpos + HALO_BLOCK_Y_MINUS) * (BLOCK_X_SIZE + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS);
}

__global__ void cukernel(
    Real *in, Real *out, Real *coeff, const IJKSize domain, const IJKSize halo, const IJKSize strides, volatile unsigned int* lock) {

    BlockSyncer blk( lock );

    unsigned int ipos, jpos;
    int iblock_pos, jblock_pos;
    const unsigned int jboundary_limit = BLOCK_Y_SIZE + HALO_BLOCK_Y_MINUS + HALO_BLOCK_Y_PLUS;
    const unsigned int iminus_limit = jboundary_limit + HALO_BLOCK_X_MINUS;
    const unsigned int iplus_limit = iminus_limit + HALO_BLOCK_X_PLUS;

    const unsigned int block_size_i =
        (blockIdx.x + 1) * BLOCK_X_SIZE < domain.m_i ? BLOCK_X_SIZE : domain.m_i - blockIdx.x * BLOCK_X_SIZE;
    const unsigned int block_size_j =
        (blockIdx.y + 1) * BLOCK_Y_SIZE < domain.m_j ? BLOCK_Y_SIZE : domain.m_j - blockIdx.y * BLOCK_Y_SIZE;

    // set the thread position by default out of the block
    iblock_pos = -HALO_BLOCK_X_MINUS - 1;
    jblock_pos = -HALO_BLOCK_Y_MINUS - 1;
    if (threadIdx.y < jboundary_limit) {
        ipos = blockIdx.x * BLOCK_X_SIZE + threadIdx.x + halo.m_i;
        jpos = blockIdx.y * BLOCK_Y_SIZE + threadIdx.y - HALO_BLOCK_Y_MINUS + halo.m_j;
        iblock_pos = threadIdx.x;
        jblock_pos = threadIdx.y - HALO_BLOCK_Y_MINUS;
    } else if (threadIdx.y < iminus_limit && threadIdx.x < BLOCK_Y_SIZE * PADDED_BOUNDARY) {
        ipos = blockIdx.x * BLOCK_X_SIZE - PADDED_BOUNDARY + threadIdx.x % PADDED_BOUNDARY + halo.m_i;
        jpos = blockIdx.y * BLOCK_Y_SIZE + threadIdx.x / PADDED_BOUNDARY + halo.m_j;
        iblock_pos = -PADDED_BOUNDARY + (int)threadIdx.x % PADDED_BOUNDARY;
        jblock_pos = threadIdx.x / PADDED_BOUNDARY;
    } else if (threadIdx.y < iplus_limit && threadIdx.x < BLOCK_Y_SIZE * PADDED_BOUNDARY) {
        ipos = blockIdx.x * BLOCK_X_SIZE + threadIdx.x % PADDED_BOUNDARY + BLOCK_X_SIZE + halo.m_i;
        jpos = blockIdx.y * BLOCK_Y_SIZE + threadIdx.x / PADDED_BOUNDARY + halo.m_j;
        iblock_pos = threadIdx.x % PADDED_BOUNDARY + BLOCK_X_SIZE;
        jblock_pos = threadIdx.x / PADDED_BOUNDARY;
    }

    int index_ = index(ipos, jpos, 0, strides);

// flx and fly can be defined with smaller cache sizes, however in order to reuse the same cache_index function, I
// defined them here
// with same size. shared memory pressure should not be too high nevertheless
#define CACHE_SIZE (BLOCK_X_SIZE + HALO_BLOCK_X_MINUS + HALO_BLOCK_X_PLUS) * (BLOCK_Y_SIZE + 2)
    __shared__ Real lap[CACHE_SIZE];
    __shared__ Real flx[CACHE_SIZE];
    __shared__ Real fly[CACHE_SIZE];

    for (int kpos = 0; kpos < domain.m_k; ++kpos) {

        if (is_in_domain< -1, 1, -1, 1 >(iblock_pos, jblock_pos, block_size_i, block_size_j)) {

            lap[cache_index(iblock_pos, jblock_pos)] =
                (Real)4 * __ldg(& in[index_] ) -
                ( __ldg(& in[index_+index(1, 0,0, strides)] ) + __ldg(& in[index_ - index(1, 0,0, strides)] ) +
                    __ldg(&in[index_+index(0, 1, 0, strides)]) + __ldg(&in[index_ - index(0, 1, 0, strides)]));
        }

        __syncthreads();
        blk.sync();

        if (is_in_domain< -1, 0, 0, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)) {
            flx[cache_index(iblock_pos, jblock_pos)] =
                lap[cache_index(iblock_pos + 1, jblock_pos)] - lap[cache_index(iblock_pos, jblock_pos)];
            if (flx[cache_index(iblock_pos, jblock_pos)] *
                    (__ldg(&in[index_+index(1, 0, 0, strides)]) - __ldg(&in[index_])) >
                0) {
                flx[cache_index(iblock_pos, jblock_pos)] = 0.;
            }
        }

        if (is_in_domain< 0, 0, -1, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)) {
            fly[cache_index(iblock_pos, jblock_pos)] =
                lap[cache_index(iblock_pos, jblock_pos + 1)] - lap[cache_index(iblock_pos, jblock_pos)];
            if (fly[cache_index(iblock_pos, jblock_pos)] *
                    (__ldg(&in[index_+index(0, 1, 0, strides)]) - __ldg(&in[index_])) >
                0) {
                fly[cache_index(iblock_pos, jblock_pos)] = 0.;
            }
        }

        __syncthreads();

        if (is_in_domain< 0, 0, 0, 0 >(iblock_pos, jblock_pos, block_size_i, block_size_j)) {
            out[index_] =
                __ldg(&in[index_]) -
                coeff[index_] *
                    (flx[cache_index(iblock_pos, jblock_pos)] - flx[cache_index(iblock_pos - 1, jblock_pos)] +
                        fly[cache_index(iblock_pos, jblock_pos)] - fly[cache_index(iblock_pos, jblock_pos - 1)]);
        }

        index_ += index(0,0,1, strides);
    }
}

void launch_kernel(repository &repo, timer_cuda* time) {
    unsigned int zero = 0;
    unsigned int* lock;
    hipMalloc( &lock, sizeof(unsigned int) );
    hipMemcpy( lock, &zero, sizeof(unsigned int), hipMemcpyHostToDevice );

    IJKSize domain = repo.domain();
    IJKSize halo = repo.halo();

    dim3 threads, blocks;
    threads.x = BLOCK_X_SIZE;
    threads.y = BLOCK_Y_SIZE + HALO_BLOCK_Y_MINUS + HALO_BLOCK_Y_PLUS + (HALO_BLOCK_X_MINUS > 0 ? 1 : 0) +
                (HALO_BLOCK_X_PLUS > 0 ? 1 : 0);
    threads.z = 1;
    blocks.x = (domain.m_i + BLOCK_X_SIZE - 1) / BLOCK_X_SIZE;
    blocks.y = (domain.m_j + BLOCK_Y_SIZE - 1) / BLOCK_Y_SIZE;
    blocks.z = 1;

    IJKSize strides;
    compute_strides(domain, halo, strides);

    Real *in = repo.field_d("u_in");
    Real *out = repo.field_d("u_out");
    Real *coeff = repo.field_d("coeff");

    if(time) time->start();
    cukernel<<< blocks, threads, 0 >>>(in, out, coeff, domain, halo, strides, lock);
    if(time) time->pause();
}
